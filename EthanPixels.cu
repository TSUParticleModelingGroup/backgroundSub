// nvcc EthanPixels.cu -o temp -lm


#include <hip/hip_runtime.h>
#include <math.h>
#include <ctype.h>
#include <stdio.h>
#include <stdlib.h>

// size of vector
#define M 4	// Number of frames
#define N 10	// Number of pixels per frame

#define BLOCK 128  // Size of blocks, best if it is a power of 2.

// Globals
int *BlockOfFrames_CPU, *BlockOfFrames_GPU;
float *MeanFrame_CPU, *MeanFrame_GPU;
float *BlockOfLogNormalFrames_GPU;
float *MeanLogNormalFrame_CPU, *MeanLogNormalFrame_GPU;
float *StdvLogNormalFrame_CPU, *StdvLogNormalFrame_GPU;

dim3 dimBlock, dimGrid;

void AllocateMemory()
{
	// This are the set of frames that will be used to generate the log normal frame
	// and the standard deviation frame
	BlockOfFrames_CPU = (int *)malloc(N*M*sizeof(int)); 
	hipMalloc((void**)&BlockOfFrames_GPU,N*M*sizeof(int));
	hipMalloc((void**)&BlockOfLogNormalFrames_GPU,N*M*sizeof(float));  
	
	// Will hold the log normal frame and the standard deviation of the frames minus the log normal
	MeanFrame_CPU = (float *)malloc(N*sizeof(float));
	MeanLogNormalFrame_CPU = (float *)malloc(N*sizeof(float));
	StdvLogNormalFrame_CPU = (float *)malloc(N*sizeof(float));
	hipMalloc((void**)&MeanFrame_GPU,N*sizeof(float));
	hipMalloc((void**)&MeanLogNormalFrame_GPU,N*sizeof(float));
	hipMalloc((void**)&StdvLogNormalFrame_GPU,N*sizeof(float));
}
	
/*
	However you get you 300,000 by 80 pixels loaded in here then CUDA will do the rest.
	This is loading the big vector from 1st 300,000 then from 2nd 300,000 and so on until frame 80.
	It may be faster to load the pixels the other way 80 first pixels then 80 second pixels and so on 300000 times.
	Test it and see.
	I just load (below) some small values to check that everything is working.
	M is the number of frames and N is the number of pixels per frame
*/
void loadPixels()
{
	for(int i = 0; i < M; i++)
	{
		for(int j = 0; j < N; j++)
		{
			BlockOfFrames_CPU[j +i*N] = i*j*5;
		}
	}
	for(int j = 0; j < N; j++)
	{
		MeanFrame_CPU[j] = -1.0;
		MeanLogNormalFrame_CPU[j] = -1.0;
		StdvLogNormalFrame_CPU[j] = -1.0;
	}
}

void SetUpCudaDevices()
{
	dimBlock.x = BLOCK;
	dimBlock.y = 1;
	dimBlock.z = 1;

	dimGrid.x = ((N-1)/BLOCK)+1;
	dimGrid.y = 1;
	dimGrid.z = 1;
}

void copyFramessUp()
{
	hipMemcpyAsync(BlockOfFrames_GPU, BlockOfFrames_CPU, N*M*sizeof(int), hipMemcpyHostToDevice);
}

__global__ void creatingMeanPixelFrame(float *meanFrame, int *allFrames, int pixelsPerFrame, int frames)
{
	int pixel = threadIdx.x + blockIdx.x*blockDim.x;
	if(pixel < pixelsPerFrame)
	{
		float sum = 0.0;
		for(int i = 0; i < frames; i++)
		{
			sum += allFrames[pixel + pixelsPerFrame*i];
		}
		meanFrame[pixel] = sum/(float)frames;
	}
}

__global__ void creatingLogNormalFrames(float *meanFrame, int *allFrames, float *allFramesLogNormal, int pixelsPerFrame, int frames)
{
	int pixel = threadIdx.x + blockIdx.x*blockDim.x;
	if(pixel < pixelsPerFrame)
	{
		for(int i = 0; i < frames; i++)
		{
			allFramesLogNormal[pixel + pixelsPerFrame*i] = (float)allFrames[pixel + pixelsPerFrame*i] -  meanFrame[pixel];
			allFramesLogNormal[pixel + pixelsPerFrame*i] = abs(allFramesLogNormal[pixel + pixelsPerFrame*i]);
			// WHat do you do if this is zero???
			if(allFramesLogNormal[pixel + pixelsPerFrame*i] == 0.0) allFramesLogNormal[pixel + pixelsPerFrame*i] = 0.000001;
			allFramesLogNormal[pixel + pixelsPerFrame*i] = logf(allFramesLogNormal[pixel + pixelsPerFrame*i]);
		}
	}
}

__global__ void creatingMeanLogNormalFrame(float *meanlogNormalFrame, float *allFramesLogNormal, int pixelsPerFrame, int frames)
{
	int pixel = threadIdx.x + blockIdx.x*blockDim.x;
	if(pixel < pixelsPerFrame)
	{
		float sum = 0.0;
		for(int i = 0; i < frames; i++)
		{
			sum += allFramesLogNormal[pixel + pixelsPerFrame*i];
		}
		meanlogNormalFrame[pixel] = sum/(float)frames;
	}
}

__global__ void creatingStdvLogNormalFrame(float *stdvLogNormalFrame, float *meanLogNormalFrame, float *allFramesLogNormal, int pixelsPerFrame, int frames)
{
	int pixel = threadIdx.x + blockIdx.x*blockDim.x;
	float temp;
	if(pixel < pixelsPerFrame)
	{
		float sum = 0.0;
		for(int i = 0; i < frames; i++)
		{
			temp = allFramesLogNormal[pixel + pixelsPerFrame*i] - meanLogNormalFrame[pixel];
			sum += temp*temp;
		}
		stdvLogNormalFrame[pixel] = sqrtf((sum*sum)/(float)(frames-1));
	}
}

void copyFramesDown()
{
	hipMemcpyAsync(MeanFrame_CPU, MeanFrame_GPU, N*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpyAsync(MeanLogNormalFrame_CPU, MeanLogNormalFrame_GPU, N*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpyAsync(StdvLogNormalFrame_CPU, StdvLogNormalFrame_GPU, N*sizeof(float), hipMemcpyDeviceToHost);
}

void stats()
{
	for(int i = 0; i < N; i++)
	{
		printf("MeanFrame_CPU[%d] = %f MeanLogNormalFrame_CPU[%d] = %f StdvLogNormalFrame_CPU[%d] = %f\n", i, MeanFrame_CPU[i], i, MeanLogNormalFrame_CPU[i], i, StdvLogNormalFrame_CPU[i]);
	}
}

void cleanUp()
{
	free(BlockOfFrames_CPU);
	free(MeanFrame_CPU);
	free(MeanLogNormalFrame_CPU);
	free(StdvLogNormalFrame_CPU);

	hipFree(BlockOfFrames_GPU);
	hipFree(BlockOfLogNormalFrames_GPU);
	hipFree(MeanFrame_GPU);
	hipFree(MeanLogNormalFrame_GPU);
	hipFree(StdvLogNormalFrame_GPU);
}

void errorCheck(const char *message)
{
	hipError_t  error;
	error = hipGetLastError();

	if(error != hipSuccess)
	{
		printf("\n CUDA ERROR: %s = %s\n", message, hipGetErrorString(error));
		exit(0);
	}
}

int main()
{
	AllocateMemory();
	SetUpCudaDevices();
	loadPixels();
	copyFramessUp();
	errorCheck("copyFramessUp");
	
	hipDeviceSynchronize();
	creatingMeanPixelFrame<<<dimGrid,dimBlock>>>(MeanFrame_GPU, BlockOfFrames_GPU, N, M);
	errorCheck("creatingMeanPixelFrame");
	
	creatingLogNormalFrames<<<dimGrid,dimBlock>>>(MeanFrame_GPU, BlockOfFrames_GPU, BlockOfLogNormalFrames_GPU, N, M);
	errorCheck("creatingLogNormalFrames");
	
	creatingMeanLogNormalFrame<<<dimGrid,dimBlock>>>(MeanLogNormalFrame_GPU, BlockOfLogNormalFrames_GPU, N, M);
	errorCheck("creatingMeanLogNormalFrame");
	
	creatingStdvLogNormalFrame<<<dimGrid,dimBlock>>>(StdvLogNormalFrame_GPU, MeanLogNormalFrame_GPU, BlockOfLogNormalFrames_GPU, N, M);
	errorCheck("creatingStdvLogNormalFrame");
	
	copyFramesDown();
	errorCheck("copyFramesDown");
	
	hipDeviceSynchronize();
	
	stats();
	cleanUp();
	printf("\n DONE \n");
}
